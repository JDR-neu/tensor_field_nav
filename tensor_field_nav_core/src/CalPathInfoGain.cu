#include "hip/hip_runtime.h"
/***
 Simple implementation of information gains for specified topological branch
 ***/
extern "C" {
#include "tensor_field_nav_core/CalPathInfoGain.h"
}
__global__ void calPathInfoGain_device(const float *pointCollection,const float *pathPoints,int *pathInfoGain_tmp,int *pathInfoGain, int pointCollectionSize,int pathPointSize){
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    if(idx< pathPointSize){
        int curPointInfoGain=0;
        for(int i=0; i<pointCollectionSize;i++){
            float deta_x=pathPoints[2*idx]-pointCollection[3*i];
            float deta_y=pathPoints[2*idx+1]-pointCollection[3*i+1];
            float deta_z=0.3-pointCollection[3*i+2];
            float dist_quad=(deta_x*deta_x+deta_y*deta_y+deta_z*deta_z);
            if(dist_quad<1)
                curPointInfoGain++;
        }
        pathInfoGain_tmp[idx]=curPointInfoGain;
    }
    __syncthreads();
    if(idx<3){
        pathInfoGain[idx]=pathInfoGain_tmp[idx];
    }else if(idx>pathPointSize-3 && idx<pathPointSize){
        pathInfoGain[idx]=pathInfoGain_tmp[idx];
    }else if(idx>=3 &&idx<=pathPointSize-3){
        int sum_infoGain=0;
        for(int i=idx-2;i<idx+3;i++){
            sum_infoGain=sum_infoGain+pathInfoGain_tmp[idx];
        }
        pathInfoGain[idx]=int(sum_infoGain/5);
    }
}

//extern "C"
void calPathInfoGain(const float *pointCollection, const float *pathPoints, int *pathInfoGain,int pointCollectionSize, int pathPointsSize){
    float *pointCollection_dev,*pathPoints_dev;
    int *pathInfoGain_dev,*pathInfoGain_tmp_dev;
    hipError_t cudaStatus= hipMalloc((void**)&pointCollection_dev, sizeof(float)*pointCollectionSize*3);
//    hipMalloc((void**)&pointCollection_dev, sizeof(float)*pointCollectionSize*3);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "memory malloc to pointCollection failed ");
        return;
    }
    cudaStatus=hipMalloc((void**)&pathPoints_dev,sizeof(float)*pathPointsSize*2);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "memory malloc to pathPoints failed");
        return;
    }

    cudaStatus=hipMalloc((void**)&pathInfoGain_dev,sizeof(int)*pathPointsSize);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "memory malloc to pathInfoGain_dev failed");
        return;
    }

    cudaStatus=hipMalloc((void**)&pathInfoGain_tmp_dev,sizeof(int)*pathPointsSize);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "memory malloc to pathInfoGain_dev failed");
        return;
    }

    cudaStatus=hipMemcpy(pointCollection_dev,pointCollection,sizeof(float)*pointCollectionSize*3,hipMemcpyHostToDevice);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "pointCollection memory host to device failed");
        return;
    }

    cudaStatus=hipMemcpy(pathPoints_dev,pathPoints, sizeof(float)*pathPointsSize*2,hipMemcpyHostToDevice);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "pathPoints memory host to device failed");
        return;
    }

    calPathInfoGain_device<<<(pathPointsSize+63)/64,64>>>(pointCollection_dev,pathPoints_dev,pathInfoGain_tmp_dev,pathInfoGain_dev,pointCollectionSize,pathPointsSize);

    cudaStatus=hipMemcpy(pathInfoGain,pathInfoGain_dev, sizeof(int)*pathPointsSize,hipMemcpyDeviceToHost);
    if(cudaStatus !=hipSuccess){
        fprintf(stderr, "pathPoints memory device to host failed");
        return;
    }

    hipFree(pointCollection_dev);
    hipFree(pathPoints_dev);
    hipFree(pathInfoGain_dev);
    hipFree(pathInfoGain_tmp_dev);
}
